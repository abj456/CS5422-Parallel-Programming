#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <sys/time.h>
#include <omp.h>

const int INF = ((1 << 30) - 1);
double getTimeStamp() {
    struct timeval tv;
    gettimeofday( &tv, NULL );
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

//======================
#define DEV_NO 0
#define BF 64
#define HALF_BF (BF / 2)
#define NUM_THREADS HALF_BF
#define STRINGIFY(x) #x
#define PRAGMA_UNROLL(x) _Pragma(STRINGIFY(unroll x))
#define addr(V, i, j) ((i) * V + (j))

hipDeviceProp_t prop;
int n, m;
int org_n;
int *Dist_host;
// int ncpus;
size_t dist_size;

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    
    org_n = n;
    if(n % BF != 0) {
        n += (BF - n % BF);
    }
    printf("# of vertex: %d\n", org_n);

    dist_size = n * n * sizeof(int);
    Dist_host = (int*)malloc(dist_size);

    #pragma omp parallel for collapse(2)
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                Dist_host[addr(n, i, j)] = 0;
            } else {
                Dist_host[addr(n, i, j)] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist_host[addr(n, pair[0], pair[1])] = pair[2];
    }
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    if (n == org_n) {
        fwrite(Dist_host, sizeof(int), org_n * org_n, outfile);
    }
    else {   
        for (int i = 0; i < org_n; ++i) {
            fwrite(&Dist_host[i * n], sizeof(int), org_n, outfile);
        }
    }
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }

__global__ void cal_phase1(int n, int *Dist, int B, int Round) {

    int thread_i = threadIdx.y;
    int thread_j = threadIdx.x;
    int global_i = Round * BF + thread_i;
    int global_j = Round * BF + thread_j;

    /* no shared memory */
    // for(int k = Round * B; k < n && k < (Round + 1) * B; ++k){
    //     Dist[global_i * n + global_j] = min(Dist[global_i * n + global_j], (Dist[global_i * n + k] + Dist[k * n + global_j]));
    //     __syncthreads();
    // }

    __shared__ int shared_block[BF][BF];
    int global_idx = addr(n, global_i, global_j);
    shared_block[thread_i][thread_j] = Dist[global_idx];
    shared_block[thread_i][thread_j + HALF_BF] = Dist[global_idx + HALF_BF];
    shared_block[thread_i + HALF_BF][thread_j] = Dist[global_idx + HALF_BF * n];
    shared_block[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[global_idx + HALF_BF * (n + 1)];
    __syncthreads();

    PRAGMA_UNROLL(BF)
    for(int k = 0; k < BF; ++k) {
        shared_block[thread_i][thread_j] = min(shared_block[thread_i][thread_j], 
                                              (shared_block[thread_i][k] + shared_block[k][thread_j]));
        
        shared_block[thread_i][thread_j + HALF_BF] = min(shared_block[thread_i][thread_j + HALF_BF],
                                                        (shared_block[thread_i][k] + shared_block[k][thread_j + HALF_BF]));

        shared_block[thread_i + HALF_BF][thread_j] = min(shared_block[thread_i + HALF_BF][thread_j],
                                                        (shared_block[thread_i + HALF_BF][k] + shared_block[k][thread_j]));

        shared_block[thread_i + HALF_BF][thread_j + HALF_BF] = min(shared_block[thread_i + HALF_BF][thread_j + HALF_BF],
                                                                  (shared_block[thread_i + HALF_BF][k] + shared_block[k][thread_j + HALF_BF]));
        // __syncthreads();
    }
    
    Dist[global_idx] = shared_block[thread_i][thread_j];
    Dist[global_idx + HALF_BF] = shared_block[thread_i][thread_j + HALF_BF];
    Dist[global_idx + HALF_BF * n] = shared_block[thread_i + HALF_BF][thread_j];
    Dist[global_idx + HALF_BF * (n + 1)] = shared_block[thread_i + HALF_BF][thread_j + HALF_BF];
}

__global__ void cal_phase2(int n, int *Dist, int B, int Round) {
    if(blockIdx.x == Round) return;

    int thread_i = threadIdx.y;
    int thread_j = threadIdx.x;

    // real index in Dist
    int global_i = Round * BF + thread_i;
    int global_j = Round * BF + thread_j;
    // horizontal computation -> i fixed
    int hz_i = global_i;
    int hz_j = blockIdx.x * BF + thread_j;
    // vertical computation -> j fixed
    int vt_i = blockIdx.x * BF + thread_i;
    int vt_j = global_j;

    /* no shared memory */
    // for(int k = Round * B; k < (Round + 1) * B && k < n; ++k) {
    //     Dist[hz_i * n + hz_j] = min(Dist[hz_i * n + hz_j], (Dist[hz_i * n + k] + Dist[k * n + hz_j]));
    //     Dist[vt_i * n + vt_j] = min(Dist[vt_i * n + vt_j], (Dist[vt_i * n + k] + Dist[k * n + vt_j]));
    //     __syncthreads();
    // }

    __shared__ int shared_pivot[BF][BF];
    __shared__ int shared_hz[BF][BF];
    __shared__ int shared_vt[BF][BF];

    int global_idx = addr(n, global_i, global_j);
    shared_pivot[thread_i][thread_j] = Dist[global_idx];
    shared_pivot[thread_i][thread_j + HALF_BF] = Dist[global_idx + HALF_BF];
    shared_pivot[thread_i + HALF_BF][thread_j] = Dist[global_idx + HALF_BF * n];
    shared_pivot[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[global_idx + HALF_BF * (n + 1)];

    int hz_idx = addr(n, hz_i, hz_j);
    shared_hz[thread_i][thread_j] = Dist[hz_idx];
    shared_hz[thread_i][thread_j + HALF_BF] = Dist[hz_idx + HALF_BF];
    shared_hz[thread_i + HALF_BF][thread_j] = Dist[hz_idx + HALF_BF * n];
    shared_hz[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[hz_idx + HALF_BF * (n + 1)];

    int vt_idx = addr(n, vt_i, vt_j);
    shared_vt[thread_i][thread_j] = Dist[vt_idx];
    shared_vt[thread_i][thread_j + HALF_BF] = Dist[vt_idx + HALF_BF];
    shared_vt[thread_i + HALF_BF][thread_j] = Dist[vt_idx + HALF_BF * n];
    shared_vt[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[vt_idx + HALF_BF * (n + 1)];
    
    __syncthreads();
    
    // #pragma unroll BF
    PRAGMA_UNROLL(BF)
    for(int k = 0; k < BF; ++k) {
        shared_hz[thread_i][thread_j] = min(shared_hz[thread_i][thread_j], 
                                            shared_pivot[thread_i][k] + shared_hz[k][thread_j]);
        shared_hz[thread_i][thread_j + HALF_BF] = min(shared_hz[thread_i][thread_j + HALF_BF], 
                                                      shared_pivot[thread_i][k] + shared_hz[k][thread_j + HALF_BF]);
        shared_hz[thread_i + HALF_BF][thread_j] = min(shared_hz[thread_i + HALF_BF][thread_j], 
                                                      shared_pivot[thread_i + HALF_BF][k] + shared_hz[k][thread_j]);
        shared_hz[thread_i + HALF_BF][thread_j + HALF_BF] = min(shared_hz[thread_i + HALF_BF][thread_j + HALF_BF], 
                                                                shared_pivot[thread_i + HALF_BF][k] + shared_hz[k][thread_j + HALF_BF]);
    
        shared_vt[thread_i][thread_j] = min(shared_vt[thread_i][thread_j],
                                            shared_vt[thread_i][k] + shared_pivot[k][thread_j]);
        shared_vt[thread_i][thread_j + HALF_BF] = min(shared_vt[thread_i][thread_j + HALF_BF],
                                                      shared_vt[thread_i][k] + shared_pivot[k][thread_j + HALF_BF]);
        shared_vt[thread_i + HALF_BF][thread_j] = min(shared_vt[thread_i + HALF_BF][thread_j],
                                                      shared_vt[thread_i + HALF_BF][k] + shared_pivot[k][thread_j]);
        shared_vt[thread_i + HALF_BF][thread_j + HALF_BF] = min(shared_vt[thread_i + HALF_BF][thread_j + HALF_BF],
                                                                shared_vt[thread_i + HALF_BF][k] + shared_pivot[k][thread_j + HALF_BF]);
        // __syncthreads();
    }
    
    Dist[hz_idx] = shared_hz[thread_i][thread_j];
    Dist[hz_idx + HALF_BF] = shared_hz[thread_i][thread_j + HALF_BF];
    Dist[hz_idx + HALF_BF * n] = shared_hz[thread_i + HALF_BF][thread_j];
    Dist[hz_idx + HALF_BF * (n + 1)] = shared_hz[thread_i + HALF_BF][thread_j + HALF_BF];

    Dist[vt_idx] = shared_vt[thread_i][thread_j];
    Dist[vt_idx + HALF_BF] = shared_vt[thread_i][thread_j + HALF_BF];
    Dist[vt_idx + HALF_BF * n] = shared_vt[thread_i + HALF_BF][thread_j];
    Dist[vt_idx + HALF_BF * (n + 1)] = shared_vt[thread_i + HALF_BF][thread_j + HALF_BF];
}

__global__ void cal_phase3(int n, int *Dist, int B, int Round) {
    if(blockIdx.x == Round || blockIdx.y == Round) return; // skip pivot, pivot hz, pivot vt

    __shared__ int block[BF][BF];
    __shared__ int vt[BF][BF];
    __shared__ int hz[BF][BF];

    int thread_i = threadIdx.y;
    int thread_j = threadIdx.x;
    int b_i = blockIdx.y;
    int b_j = blockIdx.x;

    /* idx of block waiting to be computed */
    int block_i = b_i * BF + thread_i;
    int block_j = b_j * BF + thread_j;

    /* idx of pivot hz block */
    int hz_i = Round * BF + thread_i; // row fixed, col changed
    int hz_j = block_j;

    /* idx of pivot vt block */
    int vt_i = block_i;
    int vt_j = Round * BF + thread_j; // row changed, col fixed
    
    /* no shared memory */
    // for(int k = Round * B; k < (Round + 1) * B && k < n; ++k) {
    //     Dist[block_i * n + block_j] = min(Dist[block_i * n + block_j], (Dist[block_i * n + k] + Dist[k * n + block_j]));
    //     __syncthreads();
    // }

    int block_idx = addr(n, block_i, block_j);
    block[thread_i][thread_j] = Dist[block_idx];
    block[thread_i][thread_j + HALF_BF] = Dist[block_idx + HALF_BF];
    block[thread_i + HALF_BF][thread_j] = Dist[block_idx + HALF_BF * n];
    block[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[block_idx + HALF_BF * (n + 1)];
    
    // row changed, col fixed
    int vt_idx = addr(n, vt_i, vt_j);
    vt[thread_i][thread_j] = Dist[vt_idx];
    vt[thread_i][thread_j + HALF_BF] = Dist[vt_idx + HALF_BF];
    vt[thread_i + HALF_BF][thread_j] = Dist[vt_idx + HALF_BF * n];
    vt[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[vt_idx + HALF_BF * (n + 1)];

    // row fixed, col changed
    int hz_idx = addr(n, hz_i, hz_j);
    hz[thread_i][thread_j] = Dist[hz_idx];
    hz[thread_i][thread_j + HALF_BF] = Dist[hz_idx + HALF_BF];
    hz[thread_i + HALF_BF][thread_j] = Dist[hz_idx + HALF_BF * n];
    hz[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[hz_idx + HALF_BF * (n + 1)];
    __syncthreads();
    
    // #pragma unroll BF
    PRAGMA_UNROLL(BF)
    for(int k = 0; k < BF; ++k) {
        block[thread_i][thread_j] = min(block[thread_i][thread_j], 
                                        vt[thread_i][k] + hz[k][thread_j]);

        block[thread_i][thread_j + HALF_BF] = min(block[thread_i][thread_j + HALF_BF], 
                                                  vt[thread_i][k] + hz[k][thread_j + HALF_BF]);

        block[thread_i + HALF_BF][thread_j] = min(block[thread_i + HALF_BF][thread_j], 
                                                  vt[thread_i + HALF_BF][k] + hz[k][thread_j]);

        block[thread_i + HALF_BF][thread_j + HALF_BF] = min(block[thread_i + HALF_BF][thread_j + HALF_BF], 
                                                            vt[thread_i + HALF_BF][k] + hz[k][thread_j + HALF_BF]);
        // __syncthreads();
    }

    Dist[block_idx] = block[thread_i][thread_j];
    Dist[block_idx + HALF_BF] = block[thread_i][thread_j + HALF_BF];
    Dist[block_idx + HALF_BF * n] = block[thread_i + HALF_BF][thread_j];
    Dist[block_idx + HALF_BF * (n + 1)] = block[thread_i + HALF_BF][thread_j + HALF_BF];
}

inline void block_FW(int n, int B, int *Dist) {
    // int round = ceil(n, B);
    int round = n / BF;
    dim3 threadsPerBlock(NUM_THREADS, NUM_THREADS);
    dim3 blocksPerGrid(round, round);
    dim3 blocks(1, round);

    for(int r = 0; r < round; ++r) {
        // printf("%d %d\n", r, round);
        // fflush(stdout);
        /* Phase 1*/
        // cal<<<1, threadsPerBlock>>>(n, Dist, B, r, r, r, 1, 1);
        cal_phase1<<<1, threadsPerBlock>>>(n, Dist, BF, r);

        /* Phase 2*/
        // cal<<<1, threadsPerBlock>>>(n, Dist, B, r, r, 0, r, 1);
        // cal<<<1, threadsPerBlock>>>(n, Dist, B, r, r, r + 1, round - r - 1, 1);
        // cal<<<1, threadsPerBlock>>>(n, Dist, B, r, 0, r, 1, r);
        // cal<<<1, threadsPerBlock>>>(n, Dist, B, r, r + 1, r, 1, round - r - 1);
        cal_phase2<<<round, threadsPerBlock>>>(n, Dist, BF, r);

        /* Phase 3*/
        // cal<<<1, threadsPerBlock>>>(n, Dist, B, r, 0, 0, r, r);
        // cal<<<1, threadsPerBlock>>>(n, Dist, B, r, 0, r + 1, round - r - 1, r);
        // cal<<<1, threadsPerBlock>>>(n, Dist, B, r, r + 1, 0, r, round - r - 1);
        // cal<<<1, threadsPerBlock>>>(n, Dist, B, r, r + 1, r + 1, round - r - 1, round - r - 1);
        cal_phase3<<<blocksPerGrid, threadsPerBlock>>>(n, Dist, BF, r);
    }
}

int main(int argc, char* argv[]) {
    int B = BF;
    /* detect how many CPUs are available */
    // cpu_set_t cpu_set;
    // sched_getaffinity(0, sizeof(cpu_set), &cpu_set);
    // printf("%d cpus available\n", CPU_COUNT(&cpu_set));
    // int ncpus = CPU_COUNT(&cpu_set);

    hipGetDeviceProperties(&prop, DEV_NO);
    // printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    double total_IO_time = 0.0;

    double before = getTimeStamp();
    input(argv[1]);
    double after = getTimeStamp();
    total_IO_time += (after - before);
    assert(Dist_host != nullptr);

    hipHostRegister(Dist_host, dist_size, hipHostRegisterDefault);

    int *Dist_dev;
    hipMalloc(&Dist_dev, dist_size);
    hipMemcpy(Dist_dev, Dist_host, dist_size, hipMemcpyHostToDevice);
    assert(Dist_dev != nullptr);

    // block_FW(B);
    block_FW(n, B, Dist_dev);
    
    hipMemcpy(Dist_host, Dist_dev, dist_size, hipMemcpyDeviceToHost);

    before = getTimeStamp();
    output(argv[2]);
    after = getTimeStamp();
    total_IO_time += (after - before);
    printf("total IO time: %lf\n", total_IO_time);

    hipHostFree(Dist_host);
    hipFree(Dist_dev);
    return 0;
}