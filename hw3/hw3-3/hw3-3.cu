#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <sys/time.h>
#include <omp.h>

const int INF = ((1 << 30) - 1);
double getTimeStamp() {
    struct timeval tv;
    gettimeofday( &tv, NULL );
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

//======================
#define DEV_NO 0
#define BF 64
#define HALF_BF 32
#define NUM_THREADS 32
#define STRINGIFY(x) #x
#define PRAGMA_UNROLL(x) _Pragma(STRINGIFY(unroll x))
#define addr(V, i, j) ((i) * V + (j))

hipDeviceProp_t prop;
int n, m;
int org_n;
int *Dist_host;
int ncpus;
size_t dist_size;

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    
    org_n = n;
    if(n % BF != 0) {
        n += (BF - n % BF);
    }
    printf("# of vertex: %d\n", org_n);

    dist_size = n * n * sizeof(int);
    Dist_host = (int*)malloc(dist_size);

    #pragma omp parallel for collapse(2)
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                Dist_host[addr(n, i, j)] = 0;
            } else {
                Dist_host[addr(n, i, j)] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist_host[addr(n, pair[0], pair[1])] = pair[2];
    }
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    if (n == org_n) {
        fwrite(Dist_host, sizeof(int), org_n * org_n, outfile);
    }
    else {   
        for (int i = 0; i < org_n; ++i) {
            fwrite(&Dist_host[i * n], sizeof(int), org_n, outfile);
        }
    }
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }

__global__ void cal_phase1(int n, int *Dist, int B, int Round) {

    int thread_i = threadIdx.y;
    int thread_j = threadIdx.x;
    int global_i = Round * BF + thread_i;
    int global_j = Round * BF + thread_j;

    __shared__ int shared_block[BF][BF];
    int global_idx0 = addr(n, global_i, global_j);
    int global_idx1 = global_idx0 + HALF_BF * n;
    int global_idx2 = global_idx0 + HALF_BF;
    int global_idx3 = global_idx0 + HALF_BF * (n + 1);
    shared_block[thread_i][thread_j] = Dist[global_idx0];
    shared_block[thread_i + HALF_BF][thread_j] = Dist[global_idx1];
    shared_block[thread_i][thread_j + HALF_BF] = Dist[global_idx2];
    shared_block[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[global_idx3];
    __syncthreads();

    /* no shared memory */
    // for(int k = Round * B; k < n && k < (Round + 1) * B; ++k){
    //     Dist[global_i * n + global_j] = min(Dist[global_i * n + global_j], (Dist[global_i * n + k] + Dist[k * n + global_j]));
    //     __syncthreads();
    // }

    PRAGMA_UNROLL(BF)
    for(int k = 0; k < BF; ++k) {
        shared_block[thread_i][thread_j] = min(shared_block[thread_i][thread_j], 
                                              (shared_block[thread_i][k] + shared_block[k][thread_j]));
        
        shared_block[thread_i + HALF_BF][thread_j] = min(shared_block[thread_i + HALF_BF][thread_j],
                                                        (shared_block[thread_i + HALF_BF][k] + shared_block[k][thread_j]));

        shared_block[thread_i][thread_j + HALF_BF] = min(shared_block[thread_i][thread_j + HALF_BF],
                                                        (shared_block[thread_i][k] + shared_block[k][thread_j + HALF_BF]));

        shared_block[thread_i + HALF_BF][thread_j + HALF_BF] = min(shared_block[thread_i + HALF_BF][thread_j + HALF_BF],
                                                                  (shared_block[thread_i + HALF_BF][k] + shared_block[k][thread_j + HALF_BF]));
        // __syncthreads();
    }
    
    Dist[global_idx0] = shared_block[thread_i][thread_j];
    Dist[global_idx1] = shared_block[thread_i + HALF_BF][thread_j];
    Dist[global_idx2] = shared_block[thread_i][thread_j + HALF_BF];
    Dist[global_idx3] = shared_block[thread_i + HALF_BF][thread_j + HALF_BF];
}

__global__ void cal_phase2(int n, int *Dist, int B, int Round) {
    if(blockIdx.x == Round) return;

    int thread_i = threadIdx.y;
    int thread_j = threadIdx.x;

    // real index in Dist
    int global_i = Round * BF + thread_i;
    int global_j = Round * BF + thread_j;
    // horizontal computation -> i fixed
    int hz_i = global_i;
    int hz_j = blockIdx.x * BF + thread_j;
    // vertical computation -> j fixed
    int vt_i = blockIdx.x * BF + thread_i;
    int vt_j = global_j;

    __shared__ int shared_pivot[BF][BF];
    __shared__ int shared_hz[BF][BF];
    __shared__ int shared_vt[BF][BF];

    int global_idx = addr(n, global_i, global_j);
    shared_pivot[thread_i][thread_j] = Dist[global_idx];
    shared_pivot[thread_i + HALF_BF][thread_j] = Dist[global_idx + HALF_BF * n];
    shared_pivot[thread_i][thread_j + HALF_BF] = Dist[global_idx + HALF_BF];
    shared_pivot[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[global_idx + HALF_BF * (n + 1)];

    int hz_idx = addr(n, hz_i, hz_j);
    shared_hz[thread_i][thread_j] = Dist[hz_idx];
    shared_hz[thread_i + HALF_BF][thread_j] = Dist[hz_idx + HALF_BF * n];
    shared_hz[thread_i][thread_j + HALF_BF] = Dist[hz_idx + HALF_BF];
    shared_hz[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[hz_idx + HALF_BF * (n + 1)];

    int vt_idx = addr(n, vt_i, vt_j);
    shared_vt[thread_i][thread_j] = Dist[vt_idx];
    shared_vt[thread_i + HALF_BF][thread_j] = Dist[vt_idx + HALF_BF * n];
    shared_vt[thread_i][thread_j + HALF_BF] = Dist[vt_idx + HALF_BF];
    shared_vt[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[vt_idx + HALF_BF * (n + 1)];
    
    __syncthreads();
    
    /* no shared memory */
    // for(int k = Round * B; k < (Round + 1) * B && k < n; ++k) {
    //     Dist[hz_i * n + hz_j] = min(Dist[hz_i * n + hz_j], (Dist[hz_i * n + k] + Dist[k * n + hz_j]));
    //     Dist[vt_i * n + vt_j] = min(Dist[vt_i * n + vt_j], (Dist[vt_i * n + k] + Dist[k * n + vt_j]));
    //     __syncthreads();
    // }

    // #pragma unroll BF
    PRAGMA_UNROLL(BF)
    for(int k = 0; k < BF; ++k) {
        shared_hz[thread_i][thread_j] = min(shared_hz[thread_i][thread_j], 
                                            shared_pivot[thread_i][k] + shared_hz[k][thread_j]);
        shared_hz[thread_i + HALF_BF][thread_j] = min(shared_hz[thread_i + HALF_BF][thread_j], 
                                                      shared_pivot[thread_i + HALF_BF][k] + shared_hz[k][thread_j]);
        shared_hz[thread_i][thread_j + HALF_BF] = min(shared_hz[thread_i][thread_j + HALF_BF], 
                                                      shared_pivot[thread_i][k] + shared_hz[k][thread_j + HALF_BF]);
        shared_hz[thread_i + HALF_BF][thread_j + HALF_BF] = min(shared_hz[thread_i + HALF_BF][thread_j + HALF_BF], 
                                                                shared_pivot[thread_i + HALF_BF][k] + shared_hz[k][thread_j + HALF_BF]);
    
        shared_vt[thread_i][thread_j] = min(shared_vt[thread_i][thread_j],
                                            shared_vt[thread_i][k] + shared_pivot[k][thread_j]);
        shared_vt[thread_i + HALF_BF][thread_j] = min(shared_vt[thread_i + HALF_BF][thread_j],
                                                      shared_vt[thread_i + HALF_BF][k] + shared_pivot[k][thread_j]);
        shared_vt[thread_i][thread_j + HALF_BF] = min(shared_vt[thread_i][thread_j + HALF_BF],
                                                      shared_vt[thread_i][k] + shared_pivot[k][thread_j + HALF_BF]);
        shared_vt[thread_i + HALF_BF][thread_j + HALF_BF] = min(shared_vt[thread_i + HALF_BF][thread_j + HALF_BF],
                                                                shared_vt[thread_i + HALF_BF][k] + shared_pivot[k][thread_j + HALF_BF]);
        // __syncthreads();
    }
    
    Dist[hz_idx] = shared_hz[thread_i][thread_j];
    Dist[hz_idx + HALF_BF * n] = shared_hz[thread_i + HALF_BF][thread_j];
    Dist[hz_idx + HALF_BF] = shared_hz[thread_i][thread_j + HALF_BF];
    Dist[hz_idx + HALF_BF * (n + 1)] = shared_hz[thread_i + HALF_BF][thread_j + HALF_BF];

    Dist[vt_idx] = shared_vt[thread_i][thread_j];
    Dist[vt_idx + HALF_BF * n] = shared_vt[thread_i + HALF_BF][thread_j];
    Dist[vt_idx + HALF_BF] = shared_vt[thread_i][thread_j + HALF_BF];
    Dist[vt_idx + HALF_BF * (n + 1)] = shared_vt[thread_i + HALF_BF][thread_j + HALF_BF];
}

__global__ void cal_phase3(int n, int *Dist, int Round, int row_offset) {
    if(blockIdx.x == Round || blockIdx.y + row_offset == Round) return; // skip pivot, pivot hz, pivot vt

    __shared__ int block[BF][BF];
    __shared__ int vt[BF][BF];
    __shared__ int hz[BF][BF];

    int thread_i = threadIdx.y;
    int thread_j = threadIdx.x;
    int b_i = blockIdx.y + row_offset;
    int b_j = blockIdx.x;

    /* idx of block waiting to be computed */
    int block_i = b_i * BF + thread_i;
    int block_j = b_j * BF + thread_j;

    /* idx of pivot hz block */
    int hz_i = Round * BF + thread_i; // row fixed, col changed
    // int hz_j = block_j;

    /* idx of pivot vt block */
    // int vt_i = block_i;
    int vt_j = Round * BF + thread_j; // row changed, col fixed


    int block_idx = addr(n, block_i, block_j);
    block[thread_i][thread_j] = Dist[block_idx];
    block[thread_i + HALF_BF][thread_j] = Dist[block_idx + HALF_BF * n];
    block[thread_i][thread_j + HALF_BF] = Dist[block_idx + HALF_BF];
    block[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[block_idx + HALF_BF * (n + 1)];
    
    // row changed, col fixed
    int vt_idx = addr(n, block_i, vt_j);
    vt[thread_i][thread_j] = Dist[vt_idx];
    vt[thread_i + HALF_BF][thread_j] = Dist[vt_idx + HALF_BF * n];
    vt[thread_i][thread_j + HALF_BF] = Dist[vt_idx + HALF_BF];
    vt[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[vt_idx + HALF_BF * (n + 1)];

    // row fixed, col changed
    int hz_idx = addr(n, hz_i, block_j);
    hz[thread_i][thread_j] = Dist[hz_idx];
    hz[thread_i + HALF_BF][thread_j] = Dist[hz_idx + HALF_BF * n];
    hz[thread_i][thread_j + HALF_BF] = Dist[hz_idx + HALF_BF];
    hz[thread_i + HALF_BF][thread_j + HALF_BF] = Dist[hz_idx + HALF_BF * (n + 1)];

    __syncthreads();
    
    /* no shared memory */
    // for(int k = Round * B; k < (Round + 1) * B && k < n; ++k) {
    //     Dist[global_i * n + global_j] = min(Dist[global_i * n + global_j], (Dist[global_i * n + k] + Dist[k * n + global_j]));
    //     __syncthreads();
    // }

    // #pragma unroll BF
    PRAGMA_UNROLL(BF)
    for(int k = 0; k < BF; ++k) {
        block[thread_i][thread_j] = min(block[thread_i][thread_j], 
                                        vt[thread_i][k] + hz[k][thread_j]);

        block[thread_i + HALF_BF][thread_j] = min(block[thread_i + HALF_BF][thread_j], 
                                                  vt[thread_i + HALF_BF][k] + hz[k][thread_j]);

        block[thread_i][thread_j + HALF_BF] = min(block[thread_i][thread_j + HALF_BF], 
                                                  vt[thread_i][k] + hz[k][thread_j + HALF_BF]);

        block[thread_i + HALF_BF][thread_j + HALF_BF] = min(block[thread_i + HALF_BF][thread_j + HALF_BF], 
                                                            vt[thread_i + HALF_BF][k] + hz[k][thread_j + HALF_BF]);
        // __syncthreads();
    }

    Dist[block_idx] = block[thread_i][thread_j];
    Dist[block_idx + HALF_BF * n] = block[thread_i + HALF_BF][thread_j];
    Dist[block_idx + HALF_BF] = block[thread_i][thread_j + HALF_BF];
    Dist[block_idx + HALF_BF * (n + 1)] = block[thread_i + HALF_BF][thread_j + HALF_BF];
}

inline void block_FW(int n, int B, int **Dist) {
    
    // int round = ceil(n, B);
    int round = n / BF;
    dim3 threadsPerBlock(NUM_THREADS, NUM_THREADS);
    dim3 blocks(1, round);
    // dim3 blocksPerGrid(round, round);

    #pragma omp parallel num_threads(2)
    {
        unsigned int cpu_tid = omp_get_thread_num();
        unsigned int half_round = round / 2;
        unsigned int row_offset = (cpu_tid) ? half_round : 0;
        unsigned int div_row = (cpu_tid) ? (round - half_round) : half_round;
        
        hipSetDevice(cpu_tid);
        hipMalloc(&Dist[cpu_tid], dist_size);
        assert(Dist[cpu_tid] != nullptr);
        hipMemcpy(Dist[cpu_tid], Dist_host, dist_size, hipMemcpyHostToDevice);
        dim3 blocksPerGrid(round, div_row);
        
        for(int r = 0; r < round; ++r) {
            if(r >= row_offset && r < row_offset + div_row) {
                hipMemcpy(Dist[!cpu_tid] + r * BF * n, Dist[cpu_tid] + r * BF * n, BF * n * sizeof(int), hipMemcpyDeviceToDevice);
            }
            #pragma omp barrier

            /* Phase 1*/
            cal_phase1<<<1, threadsPerBlock>>>(n, Dist[cpu_tid], BF, r);
            /* Phase 2*/
            cal_phase2<<<round, threadsPerBlock>>>(n, Dist[cpu_tid], BF, r);
            /* Phase 3*/
            cal_phase3<<<blocksPerGrid, threadsPerBlock>>>(n, Dist[cpu_tid], r, row_offset);
        }
        hipMemcpy(Dist_host + row_offset * BF * n, Dist[cpu_tid] + (row_offset * BF * n), 
                   div_row * BF * n * sizeof(int), hipMemcpyDeviceToHost);
        #pragma omp barrier
        
        hipFree(Dist[cpu_tid]);
    }
}

int main(int argc, char* argv[]) {
    int B = BF;
    /* detect how many CPUs are available */
    // cpu_set_t cpu_set;
    // sched_getaffinity(0, sizeof(cpu_set), &cpu_set);
    // printf("%d cpus available\n", CPU_COUNT(&cpu_set));
    // int ncpus = CPU_COUNT(&cpu_set);

    hipGetDeviceProperties(&prop, DEV_NO);
    // printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);

    // double total_IO_time = 0.0;

    // double before = getTimeStamp();
    input(argv[1]);
    // double after = getTimeStamp();
    // total_IO_time += (after - before);
    assert(Dist_host != nullptr);

    hipHostRegister(Dist_host, dist_size, hipHostRegisterDefault);

    int *Dist_dev[2];

    // block_FW(B);
    block_FW(n, B, Dist_dev);
    
    // cudaMemcpy(Dist_host, Dist_dev, dist_size, cudaMemcpyDeviceToHost);
    // before = getTimeStamp();
    output(argv[2]);
    // after = getTimeStamp();
    // total_IO_time += (after - before);
    // printf("total IO time: %lf\n", total_IO_time);

    hipHostFree(Dist_host);
    return 0;
}